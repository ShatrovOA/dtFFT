#include <hip/hip_runtime.h>
#include <cstddef>
#include <stdio.h>


template<typename T>
__global__ void scaleKernel(T* buffer, size_t count, T scale_value)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < count) {
        buffer[idx] = buffer[idx] * scale_value;
    }
}

void launchScaleKernel(void* buffer, size_t count, size_t scale, bool is_double, hipStream_t stream)
{
    int threads = 256;
    int blocks = (count + threads - 1) / threads;
    double scale_value = 1.0 / static_cast<double>(scale);

    if ( is_double ) {
        scaleKernel<double><<<blocks, threads, 0, stream>>>(static_cast<double*>(buffer), count, scale_value);
    } else {
        scaleKernel<float><<<blocks, threads, 0, stream>>>(static_cast<float*>(buffer), count, static_cast<float>(scale_value));
    }
}

#ifdef __cplusplus
extern "C" {
#endif

// C/Fortran wrappers
void scaleFloatCUDA(void* buffer, size_t count, size_t scale, hipStream_t stream)
{
    launchScaleKernel(buffer, count, scale, 0, stream);
}

void scaleDoubleCUDA(void* buffer, size_t count, size_t scale, hipStream_t stream)
{
    launchScaleKernel(buffer, count, scale, 1, stream);
}

void scaleComplexFloatCUDA(void* buffer, size_t count, size_t scale, hipStream_t stream)
{
    scaleFloatCUDA(buffer, 2 * count, scale, stream);
}

void scaleComplexDoubleCUDA(void* buffer, size_t count, size_t scale, hipStream_t stream)
{
    scaleDoubleCUDA(buffer, 2 * count, scale, stream);
}

#ifdef __cplusplus
}
#endif
